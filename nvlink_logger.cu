/*

nvmlReturn_t nvmlDeviceGetHandleByIndex (unsigned
int index, nvmlDevice_t *device)
Parameters
index
The index of the target GPU, >= 0 and < accessibleDevices
device
Reference in which to return the device handle
Returns
‣ NVML_SUCCESS if device has been set
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
‣ NVML_ERROR_INVALID_ARGUMENT if index is invalid or device is NULL
Modules
www.nvidia.com
NVML vR384 | 47
‣ NVML_ERROR_INSUFFICIENT_POWER if any attached devices have improperly
attached external power cables
‣ NVML_ERROR_NO_PERMISSION if the user doesn't have permission to talk to this
device
‣ NVML_ERROR_IRQ_ISSUE if NVIDIA kernel detected an interrupt issue with the
attached GPUs
‣ NVML_ERROR_GPU_IS_LOST if the target GPU has fallen off the bus or is
otherwise inaccessible
‣ NVML_ERROR_UNKNOWN on any unexpected error



nvmlReturn_t nvmlDeviceGetFieldValues (nvmlDevice_t
device, int valuesCount, nvmlFieldValue_t *values)
Parameters
device
The device handle of the GPU to request field values for
valuesCount
Number of entries in values that should be retrieved
values
Array of valuesCount structures to hold field values. Each value's fieldId must be
populated prior to this call
Returns
‣ NVML_SUCCESS if any values in values were populated. Note that you must check
the nvmlReturn field of each value for each individual status
‣ NVML_ERROR_INVALID_ARGUMENT if device is invalid or values is NULL


Queryable field for nvlink counts
#define NVML_FI_DEV_NVLINK_LINK_COUNT



nvmlReturn_t nvmlDeviceGetNvLinkState (nvmlDevice_t
device, unsigned int link, nvmlEnableState_t *isActive)
Parameters
device
The identifier of the target device
link
Specifies the NvLink link to be queried
isActive
nvmlEnableState_t where NVML_FEATURE_ENABLED indicates that the link is
active and NVML_FEATURE_DISABLED indicates it is inactive
Returns
‣ NVML_SUCCESS if isActive has been set
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
Modules
www.nvidia.com
NVML vR384 | 127
‣ NVML_ERROR_INVALID_ARGUMENT if device or link is invalid or isActive is
NULL
‣ NVML_ERROR_NOT_SUPPORTED if the device doesn't support this feature
‣ NVML_ERROR_UNKNOWN on any unexpected error





nvmlReturn_t nvmlDeviceSetNvLinkUtilizationControl
(nvmlDevice_t device, unsigned int link, unsigned
int counter, nvmlNvLinkUtilizationControl_t *control,
unsigned int reset)
Parameters
device
The identifier of the target device
link
Specifies the NvLink link to be queried
counter
Specifies the counter that should be set (0 or 1).
control
A reference to the nvmlNvLinkUtilizationControl_t to set
Modules
www.nvidia.com
NVML vR384 | 131
reset
Resets the counters on set if non-zero
Returns
‣ NVML_SUCCESS if the control has been set successfully
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
‣ NVML_ERROR_INVALID_ARGUMENT if device, counter, link, or control is
invalid
‣ NVML_ERROR_NOT_SUPPORTED if the device doesn't support this feature
‣ NVML_ERROR_UNKNOWN on any unexpected error


nvmlReturn_t nvmlDeviceGetNvLinkUtilizationControl
(nvmlDevice_t device, unsigned int link, unsigned int
counter, nvmlNvLinkUtilizationControl_t *control)
Parameters
device
The identifier of the target device
link
Specifies the NvLink link to be queried
counter
Specifies the counter that should be set (0 or 1).
control
A reference to the nvmlNvLinkUtilizationControl_t to place information
Returns
‣ NVML_SUCCESS if the control has been set successfully
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
‣ NVML_ERROR_INVALID_ARGUMENT if device, counter, link, or control is
invalid
‣ NVML_ERROR_NOT_SUPPORTED if the device doesn't support this feature
‣ NVML_ERROR_UNKNOWN on any unexpected errornvmlReturn_t nvmlDeviceGetNvLinkUtilizationControl

*/


#include <hip/hip_runtime.h>
#include "nvml.h"
#include <stdio.h>
#include <unistd.h>
#include <signal.h>
#include <sys/time.h>

#define MAX_NUM_DEVICES 32

#define NVML_CHECK(error) nvml_check(error,__FILE__,__LINE__)


//This value is not defined in the nvml.h header for cuda-9.0, but is
//included in the docs...
#define NVML_FI_DEV_NVLINK_LINK_COUNT 91

//Nvlink counter to use (0 or 1)
#define COUNTER 0

void nvml_check(nvmlReturn_t error, const char * filename, unsigned int line_num) {
  if(error == NVML_SUCCESS) {
    //success
  } else {
    fprintf(stderr, "NVML error code %d in file %s line %d\n", (int) error, filename, line_num);
  }
}

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return(tv.tv_sec + (tv.tv_usec/1000000.));
}

FILE * file_handle = NULL;
bool kill_process = false;

void sig_term_handler(int signum, siginfo_t *info, void *ptr) {
  file_handle = fopen("datasets.py", "r");
  if (file_handle != NULL) fclose(file_handle);
  printf("Caught kill signal, closing data file\n");
  kill_process = true;
}

void catch_sigterm() {
  static struct sigaction _sigact;

  memset(&_sigact, 0, sizeof(_sigact));
  _sigact.sa_sigaction = sig_term_handler;
  _sigact.sa_flags = SA_SIGINFO;

  sigaction(SIGTERM, &_sigact, NULL);
  sigaction(SIGINT, &_sigact, NULL);
}


int main(int argc, char ** argv) {
  useconds_t delay = 1000000;//micro seconds
  if(argc >= 2) {
    delay = (useconds_t) (atof(argv[1]) * 1000000);
  }
  printf("Delay %d microseconds\n", delay);
  
  NVML_CHECK(nvmlInit());
  
  nvmlDevice_t devices[MAX_NUM_DEVICES];
  unsigned int num_devices = 0;
  while(nvmlDeviceGetHandleByIndex (num_devices, devices + num_devices) == NVML_SUCCESS) {
    num_devices++;
  }

  if(num_devices == 0) {
    fprintf(stderr, "Error: No devices found\n");
    nvmlShutdown();
    return(1);
  }

  unsigned int * num_links = new unsigned int[num_devices];
  
  nvmlFieldValue_t field_value;
  field_value.fieldId = NVML_FI_DEV_NVLINK_LINK_COUNT;
  unsigned int total_links = 0;
  for(int i = 0; i < num_devices; i++) {
    NVML_CHECK(nvmlDeviceGetFieldValues (devices[i], 1, &field_value));
    num_links[i] = field_value.value.uiVal;
    total_links += field_value.value.uiVal;
  }
  
  // char str[200];
  // nvmlSystemGetNVMLVersion(str, 200);
  // printf("%s\n", str);
  nvmlEnableState_t enabled_state;
  for(int gpu_i = 0; gpu_i < num_devices; gpu_i++) {
    for(int link_i = 0; link_i < num_links[gpu_i]; link_i++) {
      NVML_CHECK(nvmlDeviceGetNvLinkState(devices[gpu_i], link_i, &enabled_state));
      if(enabled_state != NVML_FEATURE_ENABLED) {
	fprintf(stderr, "Link not enabled\n");
	return(1);
      }
    }
  }

  nvmlNvLinkUtilizationControl_t control;
  control.pktfilter = NVML_NVLINK_COUNTER_PKTFILTER_ALL;//All types of packets
  control.units     = NVML_NVLINK_COUNTER_UNIT_BYTES;   //units of bytes
  for(int gpu_i = 0; gpu_i < num_devices; gpu_i++) {
    for(int link_i = 0; link_i < num_links[gpu_i]; link_i++) {
      //Set utilization counter for device gpu_i, link_i, specified
      //COUNTER, with all packets and units of bytes control, and true
      //reset counter to 0. 
      NVML_CHECK(nvmlDeviceSetNvLinkUtilizationControl(devices[gpu_i], link_i,
						       COUNTER, &control, true));
    }
  }

  file_handle = fopen("nvlink_usage.csv", "w");
  if(file_handle == NULL) {
    fprintf(stderr, "Could not open output file\n");
    return(1);
  }
  
  unsigned long long * data_row = new unsigned long long[total_links];
  int col = 0;
  unsigned long long rx, tx;
  catch_sigterm();
  double start_time = -1.0;
  double curr_time;
  while(!kill_process) {
    col = 0;
    for(int gpu_i = 0; gpu_i < num_devices; gpu_i++) {
      for(int link_i = 0; link_i < num_links[gpu_i]; link_i++) {
	NVML_CHECK(nvmlDeviceGetNvLinkUtilizationCounter(devices[gpu_i], link_i, COUNTER, &rx, &tx));
	curr_time = get_time();
	if(start_time == -1.0) {
	  start_time = curr_time;
	}
	data_row[col++] = tx;
      }
    }
    fprintf(file_handle, "%f", curr_time - start_time);
    for(int i = 0; i < total_links; i++) {
      fprintf(file_handle, ",%llu", data_row[i]);
    }
    fprintf(file_handle, "\n");
    usleep(delay);
  }
    

  delete[] data_row;
  delete[] num_links;
  nvmlShutdown();
  return(0);
}
