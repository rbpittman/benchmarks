/*

nvmlReturn_t nvmlDeviceGetHandleByIndex (unsigned
int index, nvmlDevice_t *device)
Parameters
index
The index of the target GPU, >= 0 and < accessibleDevices
device
Reference in which to return the device handle
Returns
‣ NVML_SUCCESS if device has been set
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
‣ NVML_ERROR_INVALID_ARGUMENT if index is invalid or device is NULL
Modules
www.nvidia.com
NVML vR384 | 47
‣ NVML_ERROR_INSUFFICIENT_POWER if any attached devices have improperly
attached external power cables
‣ NVML_ERROR_NO_PERMISSION if the user doesn't have permission to talk to this
device
‣ NVML_ERROR_IRQ_ISSUE if NVIDIA kernel detected an interrupt issue with the
attached GPUs
‣ NVML_ERROR_GPU_IS_LOST if the target GPU has fallen off the bus or is
otherwise inaccessible
‣ NVML_ERROR_UNKNOWN on any unexpected error



nvmlReturn_t nvmlDeviceGetFieldValues (nvmlDevice_t
device, int valuesCount, nvmlFieldValue_t *values)
Parameters
device
The device handle of the GPU to request field values for
valuesCount
Number of entries in values that should be retrieved
values
Array of valuesCount structures to hold field values. Each value's fieldId must be
populated prior to this call
Returns
‣ NVML_SUCCESS if any values in values were populated. Note that you must check
the nvmlReturn field of each value for each individual status
‣ NVML_ERROR_INVALID_ARGUMENT if device is invalid or values is NULL


Queryable field for nvlink counts
#define NVML_FI_DEV_NVLINK_LINK_COUNT



nvmlReturn_t nvmlDeviceGetNvLinkState (nvmlDevice_t
device, unsigned int link, nvmlEnableState_t *isActive)
Parameters
device
The identifier of the target device
link
Specifies the NvLink link to be queried
isActive
nvmlEnableState_t where NVML_FEATURE_ENABLED indicates that the link is
active and NVML_FEATURE_DISABLED indicates it is inactive
Returns
‣ NVML_SUCCESS if isActive has been set
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
Modules
www.nvidia.com
NVML vR384 | 127
‣ NVML_ERROR_INVALID_ARGUMENT if device or link is invalid or isActive is
NULL
‣ NVML_ERROR_NOT_SUPPORTED if the device doesn't support this feature
‣ NVML_ERROR_UNKNOWN on any unexpected error





nvmlReturn_t nvmlDeviceSetNvLinkUtilizationControl
(nvmlDevice_t device, unsigned int link, unsigned
int counter, nvmlNvLinkUtilizationControl_t *control,
unsigned int reset)
Parameters
device
The identifier of the target device
link
Specifies the NvLink link to be queried
counter
Specifies the counter that should be set (0 or 1).
control
A reference to the nvmlNvLinkUtilizationControl_t to set
Modules
www.nvidia.com
NVML vR384 | 131
reset
Resets the counters on set if non-zero
Returns
‣ NVML_SUCCESS if the control has been set successfully
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
‣ NVML_ERROR_INVALID_ARGUMENT if device, counter, link, or control is
invalid
‣ NVML_ERROR_NOT_SUPPORTED if the device doesn't support this feature
‣ NVML_ERROR_UNKNOWN on any unexpected error


nvmlReturn_t nvmlDeviceGetNvLinkUtilizationControl
(nvmlDevice_t device, unsigned int link, unsigned int
counter, nvmlNvLinkUtilizationControl_t *control)
Parameters
device
The identifier of the target device
link
Specifies the NvLink link to be queried
counter
Specifies the counter that should be set (0 or 1).
control
A reference to the nvmlNvLinkUtilizationControl_t to place information
Returns
‣ NVML_SUCCESS if the control has been set successfully
‣ NVML_ERROR_UNINITIALIZED if the library has not been successfully initialized
‣ NVML_ERROR_INVALID_ARGUMENT if device, counter, link, or control is
invalid
‣ NVML_ERROR_NOT_SUPPORTED if the device doesn't support this feature
‣ NVML_ERROR_UNKNOWN on any unexpected errornvmlReturn_t nvmlDeviceGetNvLinkUtilizationControl

*/


#include <hip/hip_runtime.h>
#include "nvml.h"
#include <stdio.h>
#include <unistd.h>
#include <signal.h>
#include <sys/time.h>
#include <vector>
using namespace std;
#define MAX_NUM_DEVICES 32
#define NUM_CONTROLS 1

#define NVML_CHECK(error) nvml_check(error,__FILE__,__LINE__)


//This value is not defined in the nvml.h header for cuda-9.0, but is
//included in the docs...
#define NVML_FI_DEV_NVLINK_LINK_COUNT 91

//Nvlink counter to use (0 or 1)
#define GET_BYTES 0
#define GET_PACKETS 1

void nvml_check(nvmlReturn_t error, const char * filename, unsigned int line_num) {
  if(error == NVML_SUCCESS) {
    //success
  } else {
    fprintf(stderr, "NVML error code %d in file %s line %d\n", (int) error, filename, line_num);
  }
}

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return(tv.tv_sec + (tv.tv_usec/1000000.));
}

FILE * file_handle = NULL;
bool kill_process = false;

void sig_term_handler(int signum, siginfo_t *info, void *ptr) {
  printf("Caught kill signal, closing data file\n");
  kill_process = true;
}

void catch_sigterm() {
  static struct sigaction _sigact;

  memset(&_sigact, 0, sizeof(_sigact));
  _sigact.sa_sigaction = sig_term_handler;
  _sigact.sa_flags = SA_SIGINFO;

  sigaction(SIGTERM, &_sigact, NULL);
  sigaction(SIGINT, &_sigact, NULL);
}

void write_header(FILE * file_handle, int num_devices, unsigned int * num_links) {
  fprintf(file_handle, "time(sec)");
  for(int gpu_i = 0; gpu_i < num_devices; gpu_i++) {
    for(int link_i = 0; link_i < num_links[gpu_i]; link_i++) {
      fprintf(file_handle, ",GPU%d_L%d", gpu_i, link_i);
    }
  }
  fprintf(file_handle, "\n");
}

//Writes times to first col, data to rest of cols. Uses num_cols for
//the length of each row in data. 
void write_data(FILE * file_handle, vector<double> & times, vector<unsigned long long *> & data, int num_cols) {
  unsigned long long * data_row;
  for(int i = 0; i < data.size(); ++i) {
    data_row = data.at(i);
    fprintf(file_handle, "%f", times.at(i));
    for(int j = 0; j < num_cols; j++) {
      fprintf(file_handle, ",%llu", data_row[j]);
    }
    fprintf(file_handle, "\n");
  }
}


  //==================//
 //==     MAIN     ==//
//==================//

int main(int argc, char ** argv) {
  useconds_t delay = 1000000;//micro seconds
  if(argc >= 2) {
    delay = (useconds_t) (atof(argv[1]) * 1000000);
  }
  printf("Delay %d microseconds\n", delay);
  
  NVML_CHECK(nvmlInit());
  
  nvmlDevice_t devices[MAX_NUM_DEVICES];
  unsigned int num_devices = 0;
  while(nvmlDeviceGetHandleByIndex (num_devices, devices + num_devices) == NVML_SUCCESS) {
    num_devices++;
  }
  //Only log first gpu
  // num_devices = 1;
  
  if(num_devices == 0) {
    fprintf(stderr, "Error: No devices found\n");
    nvmlShutdown();
    return(1);
  }

  unsigned int * num_links = new unsigned int[num_devices];
  
  nvmlFieldValue_t field_value;
  field_value.fieldId = NVML_FI_DEV_NVLINK_LINK_COUNT;
  unsigned int total_links = 0;
  for(int i = 0; i < num_devices; i++) {
    NVML_CHECK(nvmlDeviceGetFieldValues (devices[i], 1, &field_value));
    num_links[i] = field_value.value.uiVal;
    total_links += field_value.value.uiVal;
  }
  
  // char str[200];
  // nvmlSystemGetNVMLVersion(str, 200);
  // printf("%s\n", str);
  nvmlEnableState_t enabled_state;
  for(int gpu_i = 0; gpu_i < num_devices; gpu_i++) {
    for(int link_i = 0; link_i < num_links[gpu_i]; link_i++) {
      NVML_CHECK(nvmlDeviceGetNvLinkState(devices[gpu_i], link_i, &enabled_state));
      if(enabled_state != NVML_FEATURE_ENABLED) {
	fprintf(stderr, "Link not enabled\n");
	return(1);
      }
    }
  }
  
  nvmlNvLinkUtilizationControl_t controls[NUM_CONTROLS];
  controls[0].pktfilter = NVML_NVLINK_COUNTER_PKTFILTER_ALL;//All types of packets
  controls[0].units     = NVML_NVLINK_COUNTER_UNIT_BYTES;
  if(NUM_CONTROLS > 1) {
    controls[1].pktfilter = NVML_NVLINK_COUNTER_PKTFILTER_ALL;//All types of packets
    controls[1].units     = NVML_NVLINK_COUNTER_UNIT_CYCLES;
  }
  
  
  for(int control_idx = 0; control_idx < NUM_CONTROLS; control_idx++) {
    for(int gpu_i = 0; gpu_i < num_devices; gpu_i++) {
      for(int link_i = 0; link_i < num_links[gpu_i]; link_i++) {
	//Set utilization counter for device gpu_i, link_i, specified
	//COUNTER, with all packets and units of bytes control, and true
	//reset counter to 0. 
	NVML_CHECK(nvmlDeviceSetNvLinkUtilizationControl(devices[gpu_i], link_i,
							 control_idx, controls + control_idx, 1));
      }
    }
  }
  
  int col = 0;
  unsigned long long rx, tx;
  unsigned long long * data_row;
  double start_time = -1.0;
  double curr_time;
  vector<double> times;
  vector<unsigned long long *> data[NUM_CONTROLS];
  
  catch_sigterm();
  while(!kill_process) {
    curr_time = get_time();
    if(start_time == -1.0) {
      start_time = curr_time;
    }
    for(int control_idx = 0; control_idx < NUM_CONTROLS; control_idx++) {
      col = 0;
      data_row = new unsigned long long[total_links];
      for(int gpu_i = 0; gpu_i < num_devices; gpu_i++) {
	for(int link_i = 0; link_i < num_links[gpu_i]; link_i++) {
	  NVML_CHECK(nvmlDeviceGetNvLinkUtilizationCounter(devices[gpu_i], link_i, control_idx, &rx, &tx));
	  if(controls[control_idx].units == NVML_NVLINK_COUNTER_UNIT_BYTES)
	    data_row[col++] = 8 * tx;//convert bytes to bits
	  else
	    data_row[col++] = tx;
	}
      }
      data[control_idx].push_back(data_row);
    }
    times.push_back(curr_time-start_time);
    usleep(delay);
  }
  
  //Write data for control 1
  file_handle = fopen("nvlink_usage_bits.csv", "w");
  if(file_handle == NULL) {
    fprintf(stderr, "Could not open output file\n");
    return(1);
  }
  
  write_header(file_handle, num_devices, num_links);
  write_data(file_handle, times, data[0], total_links);
  fclose(file_handle);

  if(NUM_CONTROLS > 1) {
    file_handle = fopen("nvlink_usage_cycles.csv", "w");
    if(file_handle == NULL) {
      fprintf(stderr, "Could not open output file\n");
      return(1);
    }
    
    write_header(file_handle, num_devices, num_links);
    write_data(file_handle, times, data[1], total_links);
    fclose(file_handle);
  }
  
  for(int control_idx = 0; control_idx < NUM_CONTROLS; control_idx++) {
    for(int i = 0; i < data[control_idx].size(); i++) {
      delete[] data[control_idx].at(i);
    }
  }
  delete[] num_links;
  nvmlShutdown();
  return(0);
}
